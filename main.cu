#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <sys/stat.h>

#define WIDTH 1920
#define HEIGHT 1080

#include <display.h>
#include <pthread.h>
#include <math.h>
#include <operators.h>
#include <image.h>
#include <sobel.h>
#include <view.h>

#ifndef TITLE
#define TITLE "CUDA DEBAYER DEMO"
#endif

#define SYM(v, r) (min(max((int)(v), -(int)(v)), 2*(int)(r)-(int)(v)))
#define RC(type, var, pitch, x, y, width, height) ((type*)(((uint8_t*)(var)) + SYM((y),(height)) * (pitch)) + SYM((x),(width)))
#define IS_R(x,y) (~(x|y)&1)
#define IS_G(x,y) ((x^y)&1)
#define IS_B(x,y) (x&y&1)

#define Lab_e 0.008856f
#define Lab_k 903.3f
#define Lab_v 0.0031308
#define Lab_vi 0.04045

__constant__ __device__ float Lab_M[9];
__constant__ __device__ float Lab_Mi[9];
__constant__ __device__ float3 Lab_W;


__global__
void f_cielab_enhance(float3* lab, size_t pitch_in, size_t width, size_t height, float angle)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;
	
	float3* px = RC(float3, lab, pitch_in, x, y, width, height);

	px->y = cos(angle)  * px->y + sin(angle) * px->z;
	px->z = -sin(angle) * px->y + cos(angle) * px->z;
	//px->x = 1.4;
}

__global__
void f_pgm8(float4* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height, size_t scale, int dx, int dy)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;
	
	float px = *RC(uint8_t, in, pitch_in, x/scale+dx, y/scale+dy, width, height)/255.0;
	*RC(float4, out, pitch_out, x, y, width, height) = make_float4(px, px, px, 1.0);
}

__global__
void f_ppm8(float4* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height, size_t scale, int dx, int dy)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;
	uchar3 p = *RC(uchar3, in, pitch_in, x/scale+dx, y/scale+dy, width, height);
	*RC(float4, out, pitch_out, x, y, width, height) = make_float4(p.x/255.0f, p.y/255.0f, p.z/255.0f, 1.0);
}

__global__
void f_cielab(float4* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height, size_t scale, int dx, int dy)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;

	float3 p = clamp(*RC(float3, in, pitch_in, x/scale+dx, y/scale+dy, width, height)/100, -1.0f, 1.0f);
	float sat = clamp(sqrt(p.y * p.y + p.z * p.z), 0.0f, 1.0f);
	*RC(float4, out, pitch_out, x, y, width, height) = make_float4(
			p.x + p.y + p.z/2,
			p.x - p.y + p.z/2, 
			p.x - p.z, 1.0);
}
__global__
void f_ppm8_sobel_mask(float3* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{	
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;

	auto d = View2DSym<float3>(out, pitch_out, x, y, width, height);
	auto s = View2DSym<uchar3>(in, pitch_in, x, y, width, height);

	float Kx[9] = {
		+1, 0,-1,
		+2, 0,-2,
		+1, 0,-1,
	};
	float Ky[9] = {
		+1,+2,+1,
		 0, 0, 0,
		-1,-2,-1,
	};

	float3 Lx = make_float3(0,0,0);
	float3 Ly = make_float3(0,0,0);

	#pragma unroll
	for (int r=-1, i=0; r<2; r++, i++)
	{
		#pragma unroll
		for (int c=-1, j=0; c<2; c++, j++)
		{
			float  fx = Kx[i*3+j]/255.0;
			float  fy = Ky[i*3+j]/255.0;
			uchar3 ux = s(c,r);
			uchar3 uy = s(c,r);
			Lx.x += fx * ux.x;
			Lx.y += fx * ux.y;
			Lx.z += fx * ux.z;
			Ly.x += fy * uy.x;
			Ly.y += fy * uy.y;
			Ly.z += fy * uy.z;
		}
	}
#if 1
	Lx.x = Lx.y = Lx.z = (Lx.x+Lx.y+Lx.z)/3;
#endif
	float3 Lg = clamp(make_float3(
			pow(Lx.x*Lx.x + Ly.x*Ly.x, 0.5),
			pow(Lx.y*Lx.y + Ly.y*Ly.y, 0.5),
			pow(Lx.z*Lx.z + Ly.z*Ly.z, 0.5)
			), 0.0f, 1.0f);
	d(0,0) = Lg;
}

__global__
void f_ppm8_blend(
		uchar3* out, size_t pitch_out, 
		uchar3* a, size_t pitch_a, 
		uchar3* b, size_t pitch_b, 
		float3* mask, size_t pitch_mask, 
		size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;

	float3 f = *RC(float3, mask, pitch_mask, x, y, width, height);
	uchar3 va = *RC(uchar3, a, pitch_a, x, y, width, height);
	uchar3 vb = *RC(uchar3, b, pitch_b, x, y, width, height);
	float3 ia = make_float3(
			f.x * va.x / 255.0f,
			f.y * va.y / 255.0f,
			f.z * va.z / 255.0f);
	float3 ib = make_float3(
			(1-f.x) * vb.x / 255.0f,
			(1-f.y) * vb.y / 255.0f,
			(1-f.z) * vb.z / 255.0f);
	float3 blend = ia;

	*RC(uchar3, out, pitch_out, x, y, width, height) = make_uchar3(blend.x*255, blend.y*255, blend.z*255);
}

__global__
void f_ppm8_bayer_pgm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;
	uchar3  p = *RC(uchar3, in, pitch_in, x, y, width, height);
	*RC(uint8_t, out, pitch_out, x, y, width, height) = IS_R(x,y)*p.x + IS_G(x,y)*p.y + IS_B(x,y)*p.z;
}

__global__
void f_pgm8_bayer_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;
	uint8_t  p = *RC(uint8_t, in, pitch_in, x, y, width, height);
	*RC(uchar3, out, pitch_out, x, y, width, height) = make_uchar3(IS_R(x,y)*p, IS_G(x,y)*p, IS_B(x,y)*p);
}


__global__
void f_pgm8_debayer_bilinear_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
	if (x >= width || y >= height) return;
	
	auto d = View2DSym<uchar3>(out, pitch_out, x, y, width, height);
	auto s = View2DSym<uint8_t>(in,  pitch_in,  x, y, width, height);

	d(0,0) = make_uchar3(
			(s( 0, 0 )), 
			(s(-1, 0 ) + s( 1, 0 ) + s( 0,-1 ) + s( 0, 1 )) >> 2, 
			(s(-1,-1 ) + s( 1,-1 ) + s(-1, 1 ) + s( 1, 1 )) >> 2);
		
	d(1, 1) = make_uchar3(
			(s( 0, 0 ) + s( 2, 0 ) + s( 0, 2 ) + s( 2, 2 )) >> 2, 
			(s( 0, 1 ) + s( 2, 1 ) + s( 1, 0 ) + s( 1, 2 )) >> 2, 
			(s( 1, 1 )));

	d(1, 0) = make_uchar3(
			(s( 0, 0 ) + s( 2, 0 )) >> 1, 
			(s( 1, 0 )), 
			(s( 1,-1 ) + s( 1, 1 )) >> 1);

	d(0, 1) = make_uchar3(
			(s( 0, 0 ) + s( 0, 2 )) >> 1, 
			(s( 0, 1 )), 
			(s(-1, 1 ) + s( 1, 1 )) >> 1);

}

__constant__ __device__ int32_t malvar[100];
void setupMalvar(hipStream_t stream)
{
	int32_t pmalvar[100] = 
	{
		 0,  0, -2,  0,  0,
		 0,  0,  4,  0,  0,
		-2,  4,  8,  4, -2,
		 0,  0,  4,  0,  0,
		 0,  0, -2,  0,  0,

		 0,  0,  1,  0,  0,
		 0, -2,  0, -2,  0,
		-2,  8, 10,  8, -2,
		 0, -2,  0, -2,  0,
		 0,  0,  1,  0,  0,

		 0,  0, -2,  0,  0,
		 0, -2,  8, -2,  0,
		 1,  0, 10,  0,  1,
		 0, -2,  8, -2,  0,
		 0,  0, -2,  0,  0,
		
		 0,  0, -3,  0,  0,
		 0,  4,  0,  4,  0,
		-3,  0, 12,  0, -3,
		 0,  4,  0,  4,  0,
		 0,  0, -3,  0,  0,
	};
	int rc = hipMemcpyToSymbolAsync(HIP_SYMBOL(
			malvar), &pmalvar,
			100*sizeof(int32_t), 0,
			hipMemcpyHostToDevice,
			stream);

	if (hipSuccess != rc) throw "Unable to copy malvar kernels";
}

__global__
void f_pgm8_debayer_malvar_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
	if (x >= width || y >= height) return;

	auto d = View2DSym<uchar3>(out, pitch_out, x, y, width, height);
	auto s = View2DSym<uint8_t>(in, pitch_in,  x, y, width, height);
	
	int3 trr = make_int3(s(0,0), 0, 0), trg = make_int3(0, s(1,0), 0),
	     tbg = make_int3(0, s(0,1), 0), tbb = make_int3(0, 0, s(1,1));

	for (int r=-2, *m=malvar; r<3; r++)
	{
		#pragma unroll
		for (int c=-2; c<3; c++, m++)
		{
			trr.y += m[ 0] * s(c+0, r+0), trr.z += m[75] * s(c+0, r+0);
			trg.x += m[25] * s(c+1, r+0), trg.z += m[50] * s(c+1, r+0);
			tbg.x += m[50] * s(c+0, r+1), tbg.z += m[25] * s(c+0, r+1);
			tbb.x += m[75] * s(c+1, r+1), tbb.y += m[ 0] * s(c+1, r+1);
		}
	}

	d(0,0) = make_uchar3(trr.x, clamp(trr.y, 0, 0xFF0) >> 4, clamp(trr.z, 0, 0xFF0) >> 4);
	d(1,0) = make_uchar3(clamp(trg.x, 0, 0xFF0) >> 4, trg.y, clamp(trg.z, 0, 0xFF0) >> 4);
	d(0,1) = make_uchar3(clamp(tbg.x, 0, 0xFF0) >> 4, tbg.y, clamp(tbg.z, 0, 0xFF0) >> 4);
	d(1,1) = make_uchar3(clamp(tbb.x, 0 ,0xFF0) >> 4, clamp(tbb.y, 0, 0xFF0) >> 4, tbb.z);
}

__global__
void f_pgm8_debayer_nn_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x)*2;
	int y = (blockIdx.y * blockDim.y + threadIdx.y)*2;
	if (x > width || y > height) return;

	auto d = View2DSym<uchar3>(out, pitch_out, x, y, width, height);
	auto s = View2DSym<uint8_t>(in, pitch_in,  x, y, width, height);

	d(0,0) = make_uchar3(s(0,0), (s(1,0) + s(0,1)) >> 1, s(1,1));
	d(1,0) = make_uchar3(s(2,0), (s(1,0) + s(2,1)) >> 1, s(1,1));
	d(0,1) = make_uchar3(s(0,2), (s(1,2) + s(0,1)) >> 1, s(1,1));
	d(1,1) = make_uchar3(s(2,2), (s(1,2) + s(2,1)) >> 1, s(1,1));
}

__global__
void f_pgm8_debayer_adams_gg_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
	if (x >= width || y >= height) return;
	
	auto d = View2DSym<uchar3>(out, pitch_out, x, y, width, height);
	auto s = View2DSym<uint8_t>(in, pitch_in,  x, y, width, height);
	
	// greens
	d(0,0) = make_uchar3(0, 0, 0);
	d(1,0) = make_uchar3(0, s(1,0), 0);
	d(0,1) = make_uchar3(0, s(0,1), 0);
	d(1,1) = make_uchar3(0, 0, 0);

	// greens at red / blue positions
	int treshold = 1;

	#pragma unroll
	for (int i=0; i<2; i++)
	{
		float green;
		int dh = abs(s(i-1,i)-s(i+1,i))+abs(2*s(i,i)-s(i+2,i)-s(i-2,i));
		int dv = abs(s(i,i-1)-s(i,i+1))+abs(2*s(i,i)-s(i,i+2)-s(i,i-2));
		
		if (dh > dv+treshold) 
			green = (s(i,i-1)+s(i,i+1))*0.5f+(2*s(i,i)-s(i,i-2)-s(i,i+2))*0.25f;
		else if (dv > dh+treshold) 
			green = (s(i-1,i)+s(i+1,i))*0.5f+(2*s(i,i)-s(i-2,i)-s(i+2,i))*0.25f;
		else
			green = (s(i,i-1)+s(i,i+1)+s(i-1,i)+s(i+1,i))*0.25f 
			      + (4*s(i,i)-s(i,i-2)-s(i,i+2)-s(i-2,i)-s(i+2,i))*0.125f;

		d(i,i).y = (uint8_t) clamp(green, 0.0f, 255.0f);
	}
}

__global__
void f_pgm8_debayer_adams_rb_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
	if (x >= width || y >= height) return;
	
	auto d = View2DSym<uchar3>(out, pitch_out, x, y, width, height);
	auto s = View2DSym<uint8_t>(in, pitch_in,  x, y, width, height);
	
	d(0,0).x = (s(0,0));
	d(0,0).z = (s(-1,-1)+s(1,-1)+s(-1,1)+s(1,1)) >> 2;
	d(1,1).x = (s( 0, 0)+s(2, 0)+s( 0,2)+s(2,2)) >> 2;
	d(1,1).z = (s(1,1));
	d(1,0).x = (s( 0, 0)+s(2, 0)) >> 1;
	d(1,0).z = (s( 1,-1)+s(1, 1)) >> 1;
	d(0,1).x = (s( 0, 0)+s(0, 2)) >> 1;
	d(0,1).z = (s(-1, 1)+s(1, 1)) >> 1;
}

__constant__ __device__ float gunturk_h00[9];
__constant__ __device__ float gunturk_h10[9];
__constant__ __device__ float gunturk_h01[9];
__constant__ __device__ float gunturk_h11[9];
__constant__ __device__ float gunturk_g00[25];
__constant__ __device__ float gunturk_g10[25];
__constant__ __device__ float gunturk_g01[25];
__constant__ __device__ float gunturk_g11[25];
__device__ float3* gunturk_ca;
__device__ float3* gunturk_ch;
__device__ float3* gunturk_cv;
__device__ float3* gunturk_cd;
__device__ float3* gunturk_temp;
__device__ size_t gunturk_pitch;

void setupGunturk(hipStream_t stream, size_t width, size_t height)
{
	float ph0[3] = {  0.25f,   0.5f,  0.25f };
	float ph1[3] = {  0.25f,  -0.5f,  0.25f };
	float pg0[5] = { -0.125f, 0.25f,  0.75f, 0.25f, -0.125f };
	float pg1[5] = {  0.125f, 0.25f, -0.75f, 0.25f,  0.125f };

	float ph00[9],  ph10[9],  ph01[9],  ph11[9];
	float pg00[25], pg10[25], pg01[25], pg11[25];

	for (int i=0; i<3; i++) for (int j=0; j<3; j++)
	{
		ph00[i*3+j] = ph0[i] * ph0[j];
		ph10[i*3+j] = ph1[i] * ph0[j];
		ph01[i*3+j] = ph0[i] * ph1[j];
		ph11[i*3+j] = ph1[i] * ph1[j];
	}
	for (int i=0; i<5; i++) for (int j=0; j<5; j++)
	{
		pg00[i*5+j] = pg0[i] * pg0[j];
		pg10[i*5+j] = pg1[i] * pg0[j];
		pg01[i*5+j] = pg0[i] * pg1[j];
		pg11[i*5+j] = pg1[i] * pg1[j];
	}

	int rc;
	float3 *ca, *ch, *cv, *cd, *temp;
	size_t pitch;
	
	rc  = hipMallocPitch(&ca, &pitch, sizeof(float3) * width, height);
	rc |= hipMallocPitch(&ch, &pitch, sizeof(float3) * width, height);
	rc |= hipMallocPitch(&cv, &pitch, sizeof(float3) * width, height);
	rc |= hipMallocPitch(&cd, &pitch, sizeof(float3) * width, height);
	rc |= hipMallocPitch(&temp, &pitch, sizeof(float3) * width, height);
	if (hipSuccess != rc) throw "Unable to allocate gunturk intermediate buffers";
	
	rc  = hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_h00), &ph00, 9*sizeof(float), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_h10), &ph10, 9*sizeof(float), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_h01), &ph01, 9*sizeof(float), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_h11), &ph11, 9*sizeof(float), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_g00), &pg00, 25*sizeof(float), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_g10), &pg10, 25*sizeof(float), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_g01), &pg01, 25*sizeof(float), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_g11), &pg11, 25*sizeof(float), 0, hipMemcpyHostToDevice, stream);
	if (hipSuccess != rc) throw "Unable to copy gunturk filters";
	
	rc  = hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_ca), &ca, sizeof(float3*), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_ch), &ch, sizeof(float3*), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_cv), &cv, sizeof(float3*), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_cd), &cd, sizeof(float3*), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_temp), &temp, sizeof(float3*), 0, hipMemcpyHostToDevice, stream);
	rc |= hipMemcpyToSymbolAsync(HIP_SYMBOL(gunturk_pitch), &pitch, sizeof(size_t), 0, hipMemcpyHostToDevice, stream);
	if (hipSuccess != rc) throw "Unable to set gunturk intermediate buffers";
}

__global__
void f_pgm8_debayer_gunturk_gg1_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{ 
	int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
	if (x >= width || y >= height) return;

	auto ca = View2DSym<float3>(gunturk_ca, gunturk_pitch, x, y, width, height);
	auto ch = View2DSym<float3>(gunturk_ch, gunturk_pitch, x, y, width, height);
	auto cv = View2DSym<float3>(gunturk_cv, gunturk_pitch, x, y, width, height);
	auto cd = View2DSym<float3>(gunturk_cd, gunturk_pitch, x, y, width, height);
	auto s = View2DSym<uint8_t>(in, pitch_in, x, y, width, height);
	auto d = View2DSym<uchar3>(out, pitch_out, x, y, width, height);
	
	float*h00 = gunturk_h00, *h10 = gunturk_h10, *h01 = gunturk_h01, *h11 = gunturk_h11;

	ca(0,0) = ch(0,0) = cv(0,0) = cd(0,0) = make_float3(0,0,0); 
	ca(0,1) = ch(0,1) = cv(0,1) = cd(0,1) = make_float3(0,0,0); 
	ca(1,0) = ch(1,0) = cv(1,0) = cd(1,0) = make_float3(0,0,0); 
	ca(1,1) = ch(1,1) = cv(1,1) = cd(1,1) = make_float3(0,0,0); 

	for (int r=-2; r<4; r+=2)
	{
		#pragma unroll
		for (int c=-2; c<4; c+=2, h00++, h10++, h01++, h11++)
		{
			uint8_t rr = s(c, r), bb = s(c+1, r+1);
			ca(0,0).x += rr * *h00,	ca(1,1).z += bb * *h00;
			ch(0,0).x += rr * *h10,	ch(1,1).z += bb * *h10;
			cv(0,0).x += rr * *h01, cv(1,1).z += bb * *h01;
			cd(0,0).x += rr * *h11,	cd(1,1).z += bb * *h11;
			ca(0,0).y += d(c, r).y * *h00;
			ca(1,1).y += d(c+1, r+1).y * *h00;
			
		}
	}
	ch(0,0).y = ch(0,0).x, cv(0,0).y = ch(0,0).x, cd(0,0).y = cd(0,0).x;
	ch(1,1).y = ch(1,1).z, cv(1,1).y = cv(1,1).z, cd(1,1).y = cd(1,1).z;
}


__global__
void f_pgm8_debayer_gunturk_gg2_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
	int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
	if (x >= width || y >= height) return;

	auto ca = View2DSym<float3>(gunturk_ca, gunturk_pitch, x, y, width, height);
	auto ch = View2DSym<float3>(gunturk_ch, gunturk_pitch, x, y, width, height);
	auto cv = View2DSym<float3>(gunturk_cv, gunturk_pitch, x, y, width, height);
	auto cd = View2DSym<float3>(gunturk_cd, gunturk_pitch, x, y, width, height);
	auto temp = View2DSym<float3>(gunturk_temp, gunturk_pitch, x, y, width, height);
	auto s = View2DSym<uint8_t>(in, pitch_in, x, y, width, height);
	auto d = View2DSym<uchar3>(out, pitch_out, x, y, width, height);

	float *g00 = gunturk_g00, *g10 = gunturk_g10, *g01 = gunturk_h01, *g11 = gunturk_g11;
	
	temp(0,0) =  temp(1,0) = temp(0,1) = temp(1,1) = make_float3(0,0,0);

	for (int r=-4; r<6; r+=2)
	{
		#pragma unroll
		for (int c=-4; c<6; c+=2, g00++, g10++, g01++, g11++)
		{
			temp(0,0).y += ca(c,r).y * *g00
			             + ch(c,r).y * *g10
			             + cv(c,r).y * *g01
			             + cd(c,r).y * *g11;
			
			temp(1,1).y += ca(c+1,r+1).y * *g00
			             + ch(c+1,r+1).y * *g10
			             + cv(c+1,r+1).y * *g01
			             + cd(c+1,r+1).y * *g11;
		}
	}
	d(0,0).x = s(0,0);
	d(1,1).z = s(1,1);
	d(0,0).y = clamp(temp(0,0).y, 0.f, 255.f);
	d(1,1).y = clamp(temp(1,1).y, 0.f, 255.f);
}

__global__
void f_pgm8_debayer_gunturk_rb1_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{ 
	int x = (blockIdx.x * blockDim.x + threadIdx.x) ;
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;

	auto ca = &View2DSym<float3>(gunturk_ca, gunturk_pitch, x, y, width, height)(0,0);
	auto ch = &View2DSym<float3>(gunturk_ch, gunturk_pitch, x, y, width, height)(0,0);
	auto cv = &View2DSym<float3>(gunturk_cv, gunturk_pitch, x, y, width, height)(0,0);
	auto cd = &View2DSym<float3>(gunturk_cd, gunturk_pitch, x, y, width, height)(0,0);
	auto d = View2DSym<uchar3>(out, pitch_out, x, y, width, height);
	
	float *h00 = gunturk_h00, *h10 = gunturk_h10, *h01 = gunturk_h01, *h11 = gunturk_h11;

	*ca = *ch = *cv = *cd = make_float3(0,0,0); 
	
	#pragma unroll
	for (int r=-1; r<2; r++)
	{
		#pragma unroll
		for (int c=-1; c<2; c++, h00++, h10++, h01++, h11++)
		{
			uchar3 v = d(c, r);
			float3 f = make_float3(v.x, v.y, v.z);
			*ca += f * *h00;
			*ch += f * *h10;
			*cv += f * *h01;
			*cd += f * *h11;
		}
	}
	ch->x = ch->z = ch->y;
	cv->x = cv->z = cv->y;
	cd->x = cd->z = cd->y;
}

__global__
void f_pgm8_debayer_gunturk_rb2_ppm8(void* out, size_t pitch_out, void* in, size_t pitch_in, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;

	auto ca = View2DSym<float3>(gunturk_ca, gunturk_pitch, x, y, width, height);
	auto ch = View2DSym<float3>(gunturk_ch, gunturk_pitch, x, y, width, height);
	auto cv = View2DSym<float3>(gunturk_cv, gunturk_pitch, x, y, width, height);
	auto cd = View2DSym<float3>(gunturk_cd, gunturk_pitch, x, y, width, height);
	auto t = &View2DSym<float3>(gunturk_temp, gunturk_pitch, x, y, width, height)(0,0);
	auto d = &View2DSym<uchar3>(out, pitch_out, x, y, width, height)(0,0);

	float *g00 = gunturk_g00, *g10 = gunturk_g10, *g01 = gunturk_h01, *g11 = gunturk_g11;
	
	*t = make_float3(0,0,0);
	
	#pragma unroll
	for (int r=-2; r<3; r++)
	{
		#pragma unroll
		for (int c=-2; c<3; c++, g00++, g10++, g01++, g11++)
		{
			*t += ca(c,r) * *g00
			    + ch(c,r) * *g10
			    + cv(c,r) * *g01
			    + cd(c,r) * *g11;
		}
	}

	d->x = IS_R(x,y) * d->x + (1-IS_R(x,y)) * clamp(t->x, 0.0f, 255.0f);
	d->z = IS_B(x,y) * d->z + (1-IS_B(x,y)) * clamp(t->z, 0.0f, 255.0f);
}

int smToCores(int major, int minor)
{
	switch ((major << 4) | minor)
	{
		case (9999 << 4 | 9999):
			return 1;
		case 0x30:
		case 0x32:
		case 0x35:
		case 0x37:
			return 192;
		case 0x50:
		case 0x52:
		case 0x53:
			return 128;
		case 0x60:
			return 64;
		case 0x61:
		case 0x62:
			return 128;
		case 0x70:
		case 0x72:
		case 0x75:
			return 64;
		case 0x80:
		case 0x86:
			return 64;
		default:
			return 0;
	};
}

void selectGPU()
{
	int rc;
	int maxId = -1;
	uint16_t maxScore = 0;
	int count = 0;
	hipDeviceProp_t prop;

	rc = hipGetDeviceCount(&count);
	if (hipSuccess != rc) throw "hipGetDeviceCount error";
	if (count == 0) throw "No suitable cuda device found";

	for (int id = 0; id < count; id++)
	{
		rc = hipGetDeviceProperties(&prop, id);
		if (hipSuccess != rc) throw "Unable to get device properties";
		if (prop.computeMode == hipComputeModeProhibited) 
		{
			printf("GPU %d: PROHIBITED\n", id);
			continue;
		}
		int sm_per_multiproc = smToCores(prop.major, prop.minor);
		
		printf("GPU %d: \"%s\"\n", id, prop.name);
		printf(" - Compute capability: %d.%d\n", prop.major, prop.minor);
		printf(" - Multiprocessors:    %d\n", prop.multiProcessorCount);
		printf(" - SMs per processor:  %d\n", sm_per_multiproc);
		printf(" - Clock rate:         %d\n", prop.clockRate);

		uint64_t score =(uint64_t) prop.multiProcessorCount * sm_per_multiproc * prop.clockRate;
		if (score > maxScore) 
		{
			maxId = id;
			maxScore = score;
		}
	}

	if (maxId < 0) throw "All cuda devices prohibited";

	rc = hipSetDevice(maxId);
	if (hipSuccess != rc) throw "Unable to set cuda device";

	rc = hipGetDeviceProperties(&prop, maxId);
	if (hipSuccess != rc) throw "Unable to get device properties";

	printf("\nSelected GPU %d: \"%s\" with compute capability %d.%d\n\n", 
		maxId, prop.name, prop.major, prop.minor);
}

int main(int /*argc*/, char** /*argv*/)
{
	int rc;
	hipStream_t stream = 0;

	try 
	{
		printf("Selecting the best GPU\n");
		selectGPU();
		hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
		    
		dim3 blockSize = { 16, 16 };
		dim3 gridSize = { 
			(WIDTH  + blockSize.x - 1) / blockSize.x, 
			(HEIGHT + blockSize.y - 1) / blockSize.y 
		}; 
		dim3 gridSizeQ = { 
			(WIDTH/2  + blockSize.x - 1) / blockSize.x, 
			(HEIGHT/2 + blockSize.y - 1) / blockSize.y 
		}; 

		rc = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
		if (hipSuccess != rc) throw "Unable to create CUDA stream";

		auto original = Image::load("kodak.ppm");
		original->copyToDevice(stream);
		original->printInfo();
	
		auto lab = Image::create(Image::lab, original->width, original->height);
		original->toLab(lab, stream);

		auto bayer = Image::create(Image::Type::pgm, original->width, original->height);
		f_ppm8_bayer_pgm8<<<gridSize, blockSize, 0, stream>>>(
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				original->mem.device.data,
				original->mem.device.pitch,
				original->width,
				original->height
		);
		auto bayer_colored = Image::create(Image::Type::ppm, original->width, original->height);
		f_pgm8_bayer_ppm8<<<gridSize, blockSize, 0, stream>>>(
				bayer_colored->mem.device.data,
				bayer_colored->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
#if 0
		f_ppm8_to_cielab<<<gridSize, blockSize, 0, stream>>>(
				cielab, cielab_pitch, 
				(uchar3*) bayer_colored->mem.device.data,
				bayer_colored->mem.device.pitch,
				bayer_colored->width,
				bayer_colored->height
		);
#endif
		setupMalvar(stream);
		auto debayer0 = Image::create(Image::Type::ppm, original->width, original->height);
		auto debayer1 = Image::create(Image::Type::ppm, original->width, original->height);
		auto debayer2 = Image::create(Image::Type::ppm, original->width, original->height);
		auto debayer3 = Image::create(Image::Type::ppm, original->width, original->height);
		auto debayer4 = Image::create(Image::Type::ppm, original->width, original->height);
		auto debayer5 = Image::create(Image::Type::ppm, original->width, original->height);

		// NEAREST NEIGHBOR
		f_pgm8_debayer_nn_ppm8<<<gridSizeQ, blockSize, 0, stream>>>(
				debayer0->mem.device.data,
				debayer0->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
		debayer0->copyToHost(stream);
		
		// BILINEAR
		f_pgm8_debayer_bilinear_ppm8<<<gridSizeQ, blockSize, 0, stream>>>(
				debayer1->mem.device.data,
				debayer1->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
		debayer1->copyToHost(stream);
	
		// MALVAR
		f_pgm8_debayer_malvar_ppm8<<<gridSizeQ, blockSize, 0, stream>>>(
				debayer2->mem.device.data,
				debayer2->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
		debayer2->copyToHost(stream);
		
		// HAMILTON ADAMS
		f_pgm8_debayer_adams_gg_ppm8<<<gridSizeQ, blockSize, 0, stream>>>(
				debayer3->mem.device.data,
				debayer3->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
		f_pgm8_debayer_adams_rb_ppm8<<<gridSizeQ, blockSize, 0, stream>>>(
				debayer3->mem.device.data,
				debayer3->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
		debayer3->copyToHost(stream);

		// GUNTURK / ADAMS
		setupGunturk(stream, bayer->width, bayer->height);
		f_pgm8_debayer_adams_gg_ppm8<<<gridSizeQ, blockSize, 0, stream>>>(
				debayer4->mem.device.data,
				debayer4->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
		f_pgm8_debayer_adams_rb_ppm8<<<gridSizeQ, blockSize, 0, stream>>>(
				debayer4->mem.device.data,
				debayer4->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
		f_pgm8_debayer_gunturk_gg1_ppm8<<<gridSizeQ, blockSize, 0, stream>>>(
				debayer4->mem.device.data,
				debayer4->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
		
		f_pgm8_debayer_gunturk_gg2_ppm8<<<gridSizeQ, blockSize, 0, stream>>>(
				debayer4->mem.device.data,
				debayer4->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
		);
		for (int i=0; i<8; i++)
		{
			f_pgm8_debayer_gunturk_rb1_ppm8<<<gridSize, blockSize, 0, stream>>>(	
				debayer4->mem.device.data,
				debayer4->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
			);
		
			f_pgm8_debayer_gunturk_rb2_ppm8<<<gridSize, blockSize, 0, stream>>>(
				debayer4->mem.device.data,
				debayer4->mem.device.pitch,
				bayer->mem.device.data,
				bayer->mem.device.pitch,
				bayer->width,
				bayer->height
			);
		}
		debayer4->copyToHost(stream);

		// GUNTURK / MALVAR
		auto mask = Image::create(Image::Type::raw, original->width, original->height, 3, 32);

		SobelFilter sobel;
		sobel.source = original,
		sobel.destination = mask,
		sobel.run(stream);

		f_ppm8_blend<<<gridSize, blockSize, 0, stream>>>(
				(uchar3*)debayer5->mem.device.data, debayer5->mem.device.pitch,
				(uchar3*)debayer2->mem.device.data, debayer2->mem.device.pitch,
				(uchar3*)debayer4->mem.device.data, debayer4->mem.device.pitch,
				(float3*)mask->mem.device.data, mask->mem.device.pitch,
				debayer5->width, debayer5->height);

		debayer5->copyToHost(stream);


		// SETUP DISPLAY
		CudaDisplay display(TITLE, WIDTH, HEIGHT); 
		hipDeviceSynchronize();
		display.cudaMap(stream);
		
		printf("PSNR\n");
		printf("- Nearest:  %0.02f\n", debayer0->psnr(original));
		printf("- Bilinear: %0.02f\n", debayer1->psnr(original));
		printf("- Malvar:   %0.02f\n", debayer2->psnr(original));
		printf("- Adams:    %0.02f\n", debayer3->psnr(original));
		printf("- Gunturk:  %0.02f\n", debayer4->psnr(original));
		printf("- Gunturk Malvar: %0.02f\n", debayer5->psnr(original));
		printf("Creating screen\n");


		int i = 0;
		int count = 9;
		int scale = 1;
		int dx = 0, dy = 0;
		float angle = 0.04;
		Image* debayer[] = { bayer, bayer_colored, original, 
			debayer0, debayer1, debayer2, debayer3, debayer4, debayer5 };
		while (true)
		{
			f_cielab_enhance <<< gridSize, blockSize, 0, stream >>> (
				(float3*)lab->mem.device.data, lab->mem.device.pitch,
				lab->width, lab->height, angle
			);
			original->fromLab(lab, stream);
		
			f_ppm8_blend<<<gridSize, blockSize, 0, stream>>>(
				(uchar3*)original->mem.device.data, original->mem.device.pitch,
				(uchar3*)original->mem.device.data, original->mem.device.pitch,
				(uchar3*)original->mem.device.data, original->mem.device.pitch,
				(float3*)mask->mem.device.data, mask->mem.device.pitch,
				original->width, original->height);
			
			if (!i)
			{
#if 0
				f_pgm8<<<gridSize, blockSize, 0, stream>>>(
					display.CUDA.frame.data,
					display.CUDA.frame.pitch,
					bayer->mem.device.data,
					bayer->mem.device.pitch,
					bayer->width,
					bayer->height,
					scale,
					dx*scale, dy*scale
				);
#else
				f_cielab<<<gridSize, blockSize, 0, stream>>>(
					display.CUDA.frame.data,
					display.CUDA.frame.pitch,
					lab->mem.device.data,
					lab->mem.device.pitch,
					lab->width,
					lab->height,
					scale,
					dx*scale, dy*scale
				);
#endif
			}
			else
			{
				f_ppm8<<<gridSize, blockSize, 0, stream>>>(
					display.CUDA.frame.data,
					display.CUDA.frame.pitch,
					debayer[i%count]->mem.device.data,
					debayer[i%count]->mem.device.pitch,
					debayer[i%count]->width,
					debayer[i%count]->height,
					scale,
					dx*scale, dy*scale
				);
			}

			hipStreamSynchronize(stream);
			// Draw the pixelbuffer on screen
			display.cudaFinish(stream);
			display.render(stream);
		
			rc = hipGetLastError();
			if (hipSuccess != rc) throw "CUDA ERROR";

			// check escape pressed
			if (int e = display.events()) 
			{
				if (e < 0)
				{
					display.cudaUnmap(stream);
					hipStreamDestroy(stream);
					return 0;
				}
				else switch (e)
				{
					case ',': i--; if (i < 0) i=count-1; break;
					case '.': i++; if (i >= count) i=0; break;
					case '-': scale--; if (scale <= 0) scale = 1; break;
					case '=': scale++; if (scale >= 32) scale = 32; break;
					case 'w': dy+=10; break;
					case 's': dy-=10; break;
					case 'a': dx+=10; break;
					case 'd': dx-=10; break;
					case '0': 
					case '1':
					case '2':
					case '3':
					case '4':
					case '5':
					case '6':
					case '7':
					case '8':
					case '9':
						  i = e - '0';
						  break;
					default: break;
				}
			}
			usleep(100000);
		}
	}
	catch (const char* &ex)
	{
		fprintf(stderr, "ERROR: %s\n", ex);
		fflush(stderr);
	 	return 1;
	}
	return 0;
}
